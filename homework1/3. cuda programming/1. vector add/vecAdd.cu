#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCKNUM 16

__global__
void vecAdd_kernel(float *A, float *B, float *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n) C[i] = A[i] + B[i];
}

void vecAdd(float *A, float *B, float *C, int n) {
    float *dA, *dB, *dC;
    int size = n * sizeof(float);
    hipMalloc(&dA, size);
    hipMalloc(&dB, size);
    hipMalloc(&dC, size);
    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, size, hipMemcpyHostToDevice);


    vecAdd_kernel<<<(n+BLOCKNUM-1)/BLOCKNUM, BLOCKNUM>>>(dA, dB, dC, n);
    hipMemcpy(C, dC, size, hipMemcpyDeviceToHost);

    hipFree(dA); hipFree(dB); hipFree(dC); 
}